#include "hip/hip_runtime.h"
//
// Created by ian on 26.11.2021.
//


#include "../include/pclhelper.h"
#include <math.h>

void __global__
toXYZKernel(float fx,
            float fy,
            float cx,
            float cy,
            int w,
            int h,
            float* Z,
            float* z,
            float* x,
            float* y)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    int yId = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yId * w + xId;
    if (index < (w * h) && !isnan(Z[index]) && !isinf(Z[index]) && Z[index] > 0) {
        x[index] = (float(xId - cx) * Z[index] / fx) / 1000.0;
        y[index] = (float(yId - cy) * Z[index] / fy) / 1000.0;
        z[index] = Z[index] / 1000.0;
    }
}

void __global__
toXYZRGBKernel(float fx,
               float fy,
               float cx,
               float cy,
               int w,
               int h,
               float* Z,
               unsigned char* rgb,
               pcl::PointXYZRGB* points)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    int yId = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yId * w + xId;
    if (index < (w * h) && !isnan(Z[index]) && !isinf(Z[index]) && Z[index] > 0) {

        points[index].z = Z[index] * 0.001;
        points[index].x = (float(xId - cx) / fx) * points[index].z;
        points[index].y = (float(yId - cy) / fy) * points[index].z;
        int rgb_index = (3840 * yId) + (3 * xId);
        points[index].r = rgb[rgb_index];
        points[index].g = rgb[rgb_index + 1];
        points[index].b = rgb[rgb_index + 2];
    }
}

void
toXYZ(float fx,
      float fy,
      float cx,
      float cy,
      int w,
      int h,
      float* Z,
      float* z,
      float* x,
      float* y)
{
    float* gpu_Z;
    float* gpu_z;
    float* gpu_x;
    float* gpu_y;

    size_t bytes = w * h * sizeof(float);

    // Allocate memory on GPU
    hipMalloc(&gpu_Z, bytes);
    hipMalloc(&gpu_z, bytes);
    hipMalloc(&gpu_x, bytes);
    hipMalloc(&gpu_y, bytes);
    // Copy Z - depth to GPU
    hipMemcpy(gpu_Z, Z, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    dim3 blocks(threadsPerBlock, threadsPerBlock);
    dim3 grids(ceil((float)w / threadsPerBlock),
               ceil((float)h / threadsPerBlock));

    toXYZKernel<<<grids, blocks>>>(
            fx, fy, cx, cy, w, h, gpu_Z, gpu_z, gpu_x, gpu_y);

    hipMemcpy(x, gpu_x, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(y, gpu_y, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(z, gpu_z, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_Z);
    hipFree(gpu_z);
    hipFree(gpu_x);
    hipFree(gpu_y);
}

void
toXYZRGB(float fx,
         float fy,
         float cx,
         float cy,
         int w,
         int h,
         float* Z,
         unsigned char* rgb,
         pcl::PointXYZRGB* points)
{
    float* gpu_Z;
    pcl::PointXYZRGB* gpu_points;
    unsigned char* gpu_rgb;

    size_t bytes = w * h * sizeof(float);
    size_t rgb_t = 3 * w * h * sizeof(char);
    size_t gpu_t = bytes * 8;
    // Allocate memory on GPU
    hipMalloc(&gpu_Z, bytes);
    hipMalloc(&gpu_rgb, rgb_t);
    hipMalloc(&gpu_points, gpu_t);

    // Copy Z - depth to GPU
    hipMemcpy(gpu_Z, Z, bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_rgb, rgb, rgb_t, hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    dim3 blocks(threadsPerBlock, threadsPerBlock);
    dim3 grids(ceil((float)w / threadsPerBlock),
               ceil((float)h / threadsPerBlock));

    toXYZRGBKernel<<<grids, blocks>>>(
            fx, fy, cx, cy, w, h, gpu_Z, gpu_rgb, gpu_points);

    hipMemcpy(points, gpu_points, gpu_t, hipMemcpyDeviceToHost);

    hipFree(gpu_Z);
    hipFree(gpu_rgb);
    hipFree(gpu_points);
}
